#include <iostream>
#include <fstream>
#include <sstream>
#include <climits>      /* INT_MAX */
#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand */
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <omp.h>

using namespace std;

#ifndef MAX_ITER
#define MAX_ITER 500
#endif

/**
    TODO:
--1 do the convergence check thing
--2 try to use shared memory as much as you can
--3 Stanford bithacks
 */

// #define DEBUG

__host__ __device__ double dist(double x1, double y1, double x2, double y2) {
    return (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2);  // sqrt is omitted as it reduces performance.
}

__global__ void assignKernel(int *x, int *y, int *c, double *cx, double *cy, int k, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int cluster;
        double minDist = static_cast<double>(INT_MAX);
        // Assign to closest center
        for (int j = 0; j < k; j++) {
            double distance = dist(cx[j], cy[j], x[idx], y[idx]);
            if (distance < minDist) {
                minDist = distance;
                cluster = j;
            }
            // minDist = distance * static_cast<int>(distance < minDist) + minDist * static_cast<int>(!(distance < minDist));
            // cluster = j * static_cast<int>(distance < minDist) + cluster * static_cast<int>(!(distance < minDist));
        }
        c[idx] = cluster; // assign the point to the cluster with minDist
        // __threadfence();
    }
}

__global__ void updateKernel(int *x, int *y, int *c, int k, int n, double *sumx, double *sumy, int *count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        atomicAdd(&sumx[c[idx]], x[idx]);
        atomicAdd(&sumy[c[idx]], y[idx]);
        atomicAdd(&count[c[idx]], 1);
        // __syncthreads();
    }
}

__global__ void computeCentroids(int k, double *sumx, double *sumy, int *count, double *cx, double *cy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < k) {
        if (count[idx] > 0) {
            cx[idx] = sumx[idx] / count[idx];
            cy[idx] = sumy[idx] / count[idx];
        }
    }
}

void randomCenters(int *x, int *y, int n, int k, double *cx, double *cy) {
  int *centroids = new int[k];

#ifdef RANDOM
  srand (time(NULL)); //normal code
  int added = 0;
  
  while(added != k) {
    bool exists = false;
    int temp = rand() % n;
    for(int i = 0; i < added; i++) {
      if(centroids[i] == temp) {
        exists = true;
      }
    }
    if(!exists) {
      cx[added] = x[temp];
      cy[added] = y[temp];
      centroids[added++] = temp;
    }
  }
#else //deterministic init
  for(int i = 0; i < k; i++) {
     cx[i] = x[i];
     cy[i] = y[i];
     centroids[i] = i;
  }
#endif
delete[] centroids;
}

void kmeans(int *x, int *y, int *c, double *cx, double *cy, int k, int n) {
    bool end = false;
    int iter = 0;
    int * count = new int[k];

    int *d_x, *d_y, *d_c;
    double *d_cx, *d_cy, *d_sumx, *d_sumy;
    int *d_count;
    
    hipMalloc(&d_x, n * sizeof(int));
    hipMalloc(&d_y, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));
    hipMalloc(&d_cx, k * sizeof(double));
    hipMalloc(&d_cy, k * sizeof(double));
    hipMalloc(&d_sumx, k * sizeof(double));
    hipMalloc(&d_sumy, k * sizeof(double));
    hipMalloc(&d_count, k * sizeof(int));
    
    hipMemcpy(d_x, x, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cx, cx, k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_cy, cy, k * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    while (iter < MAX_ITER) {
        #ifdef DEBUG
            printf("iter %d\n", iter);
        #endif
        // Reset the sumx, sumy, and count arrays on the device
        hipMemset(d_sumx, 0, k * sizeof(double));
        hipMemset(d_sumy, 0, k * sizeof(double));
        hipMemset(d_count, 0, k * sizeof(int));

        assignKernel<<<gridSize, blockSize>>>(d_x, d_y, d_c, d_cx, d_cy, k, n);
        // cudaDeviceSynchronize();
        
        updateKernel<<<gridSize, blockSize>>>(d_x, d_y, d_c, k, n, d_sumx, d_sumy, d_count);
        // cudaDeviceSynchronize();
        
        computeCentroids<<<(k + blockSize - 1) / blockSize, blockSize>>>(k, d_sumx, d_sumy, d_count, d_cx, d_cy);
        // cudaDeviceSynchronize();

        hipMemcpy(cx, d_cx, k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(cy, d_cy, k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(count, d_count, k * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        // cudaDeviceSynchronize(); // there is one sync in main already

        #ifdef DEBUG
            for (int i = 0; i < k; i++){
                cout << "cluster " << i << " " << cx[i] << ", " << cy[i] << ", count: " << count[i] << endl;
            }
        #endif

        iter++;
    }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_c);
    hipFree(d_cx);
    hipFree(d_cy);
    hipFree(d_sumx);
    hipFree(d_sumy);
    hipFree(d_count);
}

int readfile(const string& fname, int*& x, int*& y) {
    ifstream f;
    f.open(fname.c_str());
    if (!f.is_open()) {
        cerr << "Error opening file: " << fname << endl;
        exit(-1);
    }

    string line;
    getline(f, line);  // Read the first line (number of points)
    int n = atoi(line.c_str());
    #ifdef DEBUG
        printf("file length: %d\n", n);
    #endif

    #ifdef ALIGNED_ALLOC
        x = (int*) aligned_alloc(64, n * sizeof(int));
        y = (int*) aligned_alloc(64, n * sizeof(int));
    #else
        x = new int[n];
        y = new int[n];
    #endif

    // Read the points
    int tempx, tempy;
    for (int i = 0; i < n; i++) {
        getline(f, line);
        stringstream ss(line);
        ss >> tempx >> tempy;
        x[i] = tempx;
        y[i] = tempy;
    }

    return n;
}


int main(int argc, char *argv[]) {
    // Check arguments
    if (argc - 1 != 2) {
        printf("./test <filename> <k>\n");
        exit(-1);
    }

    string fname = argv[1];
    int k = atoi(argv[2]);
    
    int *x, *y, *c;
    double *cx, *cy;

    // Read input data
    int n = readfile(fname, x, y);
    
    cx = new double[k];
    cy = new double[k];
    c = new int[n];

    // Initialize centroids
    randomCenters(x, y, n, k, cx, cy);

    // Measure k-means execution time
    double kmeans_start = omp_get_wtime();
    kmeans(x, y, c, cx, cy, k, n);

    hipDeviceSynchronize();

    double kmeans_end = omp_get_wtime();
    printf("K-Means Execution Time: %f seconds\n", kmeans_end - kmeans_start);

    #ifdef DEBUG
        for (int i = 0; i < k; i++){
            cout << "cluster " << i << " " << cx[i] << ", " << cy[i] << endl;
        }
    #endif

    double totalSSD = 0.0;
    for (int i = 0; i < n; i++) {
        int cluster = c[i];
        totalSSD += dist(x[i], y[i], cx[cluster], cy[cluster]);
    }

    printf("Sqrt of Sum of Squared Distances (SSD): %f\n", sqrt(totalSSD));

    delete[] x;
    delete[] y;
    delete[] cx;
    delete[] cy;
    delete[] c;

    return 0;
}
