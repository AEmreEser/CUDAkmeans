#include <iostream>
#include <fstream>
#include <sstream>
#include <climits>      /* INT_MAX */
#include <stdio.h>      /* printf, scanf, puts, NULL */
#include <stdlib.h>     /* srand, rand */
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <omp.h>

using namespace std;

/** 
    IMPORTANT:
        *RECOMMENDEDCONFIG: 
            FLAT = not defined, HOSTREDUCE = not defined
        *EXPLANATIONS:
            both FLAT and HOSTREDUCE result in significant performance penalties -- they will both be turned off by default
            FLAT produces wrong results for k > 1024 --> the program will automatically use regular assign kernel instead of FLAT if FLAT is specified for k > 1024
 */

#ifndef MAXITER
#define MAXITER 500
#endif

#ifndef OUTFILE
#define OUTFILE "out_cuda.txt"
#endif

#define CHECK_CUDA_ERROR() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA error: %s\n", hipGetErrorString(err)); \
        exit(-1); \
    } \
}

/** TODO:
done: --1 do the convergence check thing 
failed: --2 use n*k threads and parallelize the for loop in assignKernel -- for large k values this is the bottleneck currently
canceled: --3 Join the kernels into one big kernel and do everything on the gpu, don't lose time communicating
done:--4 Use the cuda timer stuff for measuring walltimes - not the omp walltime thing
done:-.5 use Harris reduction on convergence check
done:--6 comment cleanup
done:--7 two gpus
**/

__host__ __device__ double dist(double x1, double y1, double x2, double y2) {
    return (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2);  // sqrt is omitted as it reduces performance.
}

__device__ void atomicMin_double(double* address, double val) {
    unsigned long long* address_as_ull = (unsigned long long*)address;
    unsigned long long old = *address_as_ull;
    unsigned long long assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(min(val, __longlong_as_double(assumed))));
    } while (assumed != old);
}

struct shared_min_t {
    double dist;
    int cluster;
};

__global__ void assignKernel_flat(int *x, int *y, int *c, double *cx, double *cy, int k, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int point_idx = idx / k;  // point this thread is working on
    int cluster_idx = idx % k;  // cluster this thread is working on
    
    if (point_idx < n) {
        double distance = dist(cx[cluster_idx], cy[cluster_idx], x[point_idx], y[point_idx]);
        
        extern __shared__ shared_min_t shared_min[];
        
        int local_idx = threadIdx.x % k;
        int point_group_idx = threadIdx.x / k;

        if (local_idx == 0) {
            shared_min[point_group_idx].dist = INT_MAX;
            shared_min[point_group_idx].cluster = -1;
        }
        __syncthreads();
        
        atomicMin_double(&shared_min[point_group_idx].dist, distance);
        __syncthreads();
        
        if (distance == shared_min[point_group_idx].dist) {
            shared_min[point_group_idx].cluster = cluster_idx;
        }
        __syncthreads();
        
        if (local_idx == 0) {
            c[point_idx] = shared_min[point_group_idx].cluster;
        }
    }
}

__global__ void assignKernel(int *x, int *y, int *c, double *cx, double *cy, /* bool *changed, */ int k, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int cluster;
        double minDist = (INT_MAX); 
        // for really large values of k, the for loop creates a huge bottleneck
        for (int j = 0; j < k; j++) {
            double distance = dist(cx[j], cy[j], x[idx], y[idx]);
            if (distance < minDist) {
                minDist = distance;
                cluster = j;
            }
        }

        c[idx] = cluster; // assign the point to the cluster with minDist

    }
}

// single block, k many threads
__global__ void checkClusterChange(double *cx, double *cy, double *prev_cx, double *prev_cy, bool* changed, bool *red_change, int k) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadId > k){ return; }

    if (threadId == 0){ *red_change = false; }

    changed[threadId] = (cx[threadId] != prev_cx[threadId]) || (cy[threadId] != prev_cy[threadId]);

    __syncthreads();

    for (int s = 1; s < k; s *= 2) { 
        int index = 2 * s * threadId;
        if(index < k){ // not divergent 
            changed[threadId] |= changed[threadId + s];
        }
        __syncthreads();
    }
    // Proved suboptimal
    // for (int s = k/2; s>0; s>>=1){
    //     if (threadId < s){
    //         changed[threadId] |= changed[threadId + s];
    //     }
    //     __syncthreads();
    // }

    prev_cx[threadId] = cx[threadId];
    prev_cy[threadId] = cy[threadId];

    if (threadId == 0) {
        *red_change |= changed[0];
    }

}


__global__ void updateKernel(int *x, int *y, int *c, int k, int n, double *sumx, double *sumy, /* bool* changed, bool* cont, */ int *count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        atomicAdd(&sumx[c[idx]], x[idx]);
        atomicAdd(&sumy[c[idx]], y[idx]);
        atomicAdd(&count[c[idx]], 1);
    }
}

__global__ void computeCentroids(int k, double *sumx, double *sumy, int *count, double *cx, double *cy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < k) {
        if (count[idx] > 0) {
            cx[idx] = sumx[idx] / count[idx];
            cy[idx] = sumy[idx] / count[idx];
        } 
        #ifdef DEBUG
            else {
                printf("centroid %d: count=0\n", idx);
            }
        #endif
        // for some reason these affect performance badly, probably due to the multiplications
        // cx[idx] = (sumx[idx] / count[idx]) * (count[idx] > 0) + cx[idx] * (count[idx] <= 0);
        // cy[idx] = (sumy[idx] / count[idx]) * (count[idx] > 0) + cy[idx] * (count[idx] <= 0);
    }
}

void randomCenters(int *x, int *y, int n, int k, double *cx, double *cy) {
  int *centroids = new int[k];

#ifdef RANDOM
  srand (time(NULL)); //normal code
  int added = 0;
  
  while(added != k) {
    bool exists = false;
    int temp = rand() % n;
    for(int i = 0; i < added; i++) {
      if(centroids[i] == temp) {
        exists = true;
      }
    }
    if(!exists) {
      cx[added] = x[temp];
      cy[added] = y[temp];
      centroids[added++] = temp;
    }
  }
#else //deterministic init
  for(int i = 0; i < k; i++) {
     cx[i] = x[i];
     cy[i] = y[i];
     centroids[i] = i;
  }
#endif
delete[] centroids;
}

void writeClusterAssignments(const int* x, const int* y, const int* c, int n, const string& filename) {
    ofstream outFile(filename);
    if (!outFile) {
        throw runtime_error("Could not open file: " + filename);
    }
    
    for (int i = 0; i < n; i++) {
        outFile << x[i] << " " << y[i] << " " << c[i] << "\n";
    }
    
    outFile.close();
}

void kmeans(int *x, int *y, int *c, double *cx, double *cy, int k, int n) {
    hipEvent_t start, stop, overall_start, overall_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&overall_start);
    hipEventCreate(&overall_stop);
    hipEventRecord(overall_start, 0);

    hipSetDevice(0); // titan X

    int iter = 0;
    int * count = new int[k];
    double acc_red_time = 0.0f, acc_assign_time = 0.0f, acc_update_time = 0.0f, acc_centroid_time = 0.0f;

    int *d_x, *d_y, *d_c;
    double *d_cx, *d_cy, *d_sumx, *d_sumy; 
    int *d_count;
    bool cont;
    
    double *d_prev_cx, *d_prev_cy;
    bool *d_changed;
    bool *d_red_change;
    hipMalloc(&d_red_change, 1 * sizeof(bool));
    hipMalloc(&d_prev_cx, k * sizeof(double));
    hipMalloc(&d_prev_cy, k * sizeof(double));
    hipMalloc(&d_changed, k * sizeof(bool));

    hipMemset(d_changed, false, k*sizeof(bool));
    hipMemset(d_red_change,    false, 1*sizeof(bool));

    hipMalloc(&d_x, n * sizeof(int));
    hipMalloc(&d_y, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));
    hipMalloc(&d_cx, k * sizeof(double));
    hipMalloc(&d_cy, k * sizeof(double));
    hipMalloc(&d_sumx, k * sizeof(double));
    hipMalloc(&d_sumy, k * sizeof(double));
    hipMalloc(&d_count, k * sizeof(int));
    
    hipMemcpy(d_x, x, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cx, cx, k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_cy, cy, k * sizeof(double), hipMemcpyHostToDevice);
    printf("Begin\n");

    #ifdef FLAT // WARNING: for large values of k assignKernel_flat returns wrong results due to contention over shared memory and other stuff... Don't use it!
        if (k <= 1024){
            printf("Flat\n");
        } else { 
            printf("Warning: cannot use FLAT with k > 1024, usign regular assign kernel instead\n");
        }
    #endif

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    hipMemset(d_c, -1, n * sizeof(int)); 

    while (iter < MAXITER) {
        #ifdef DEBUG
            printf("==========%d=========\n", iter);
        #endif
        cont = false;
        
        hipMemset(d_sumx,  0, k * sizeof(double));
        hipMemset(d_sumy,  0, k * sizeof(double));
        hipMemset(d_count, 0, k * sizeof(int));

        hipEventRecord(start, 0);

        #ifdef FLAT // WARNING: for large values of k assignKernel_flat returns wrong results due to contention over shared memory and other stuff... Don't use it!
            if (k <= 1024){
                blockSize = k;
               assignKernel_flat<<<((n*k)+blockSize - 1)/blockSize, blockSize, k * sizeof(shared_min_t)>>>(d_x, d_y, d_c, d_cx, d_cy, k, n);
            } else {
                assignKernel<<<gridSize, blockSize>>>(d_x, d_y, d_c, d_cx, d_cy, k, n);
            }
        #else
            assignKernel<<<gridSize, blockSize>>>(d_x, d_y, d_c, d_cx, d_cy, k, n);
        #endif

        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float timer_assign;
        hipEventElapsedTime(&timer_assign, start, stop);
        acc_assign_time += timer_assign;
        
        hipEventRecord(start, 0);
        updateKernel<<<gridSize, blockSize>>>(d_x, d_y, d_c, k, n, d_sumx, d_sumy, d_count);
        hipDeviceSynchronize(); // wait for gpu
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float timer_update;
        hipEventElapsedTime(&timer_update, start, stop);
        acc_update_time += timer_update;
        
        hipEventRecord(start, 0);
        computeCentroids<<<(k + blockSize - 1) / blockSize, blockSize>>>(k, d_sumx, d_sumy, d_count, d_cx, d_cy);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float timer_centroid;
        hipEventElapsedTime(&timer_centroid, start, stop);
        acc_centroid_time += timer_centroid;

        hipEventRecord(start, 0);
        checkClusterChange<<<k, 1>>>(d_cx, d_cy, d_prev_cx, d_prev_cy, d_changed, d_red_change, k);
        hipDeviceSynchronize();
        hipMemcpy(&cont, d_red_change, 1 *sizeof(bool), hipMemcpyDeviceToHost);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float timer_reduce;
        hipEventElapsedTime(&timer_reduce, start, stop);
        acc_red_time += timer_reduce;


        #ifdef DEBUG
            #ifndef HOSTREDUCE // NOT defiend
                cudaMemcpy(cx, d_cx, k * sizeof(double), cudaMemcpyDeviceToHost);
                cudaMemcpy(cy, d_cy, k * sizeof(double), cudaMemcpyDeviceToHost);
            #endif
            cudaMemcpy(count, d_count, k * sizeof(int), cudaMemcpyDeviceToHost);
            cudaMemcpy(c, d_c, n * sizeof(int), cudaMemcpyDeviceToHost);
            printf("End of itertion %d-- results\n",iter);
            for (int i = 0; i < k; i++){
                cout << "cluster " << i << ": (" << cx[i] << ", " << cy[i] << ") , count: " << count[i] << endl;
            }
        #endif

        if (cont == false){ 
            #ifndef HOSTREDUCE // NOT defiend
                hipMemcpy(cx, d_cx, k * sizeof(double), hipMemcpyDeviceToHost);
                hipMemcpy(cy, d_cy, k * sizeof(double), hipMemcpyDeviceToHost);
            #endif
            hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
            printf("Converged at iteration %d\n", iter);
            writeClusterAssignments(x, y, c, n, OUTFILE);
            
            break; 
        } // means no changes -- converged

        iter++;
    }

        printf("Acc Assign: %f seconds\n", acc_assign_time/1000);


    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_c);
    hipFree(d_cx);
    hipFree(d_cy);
    hipFree(d_prev_cx);
    hipFree(d_prev_cy);
    hipFree(d_sumx);
    hipFree(d_changed);
    hipFree(d_red_change);
    hipFree(d_sumy);
    hipFree(d_count);

    hipEventRecord(overall_stop, 0);
    hipEventSynchronize(overall_stop);
    float timer_overall;
    hipEventElapsedTime(&timer_overall, overall_start, overall_stop);
    printf("Kmeans total runtime: %f seconds (Cuda events)\n", timer_overall/1000);
}

void kmeans_multigpu(int *x, int *y, int *c, double *cx, double *cy, int k, int n) {
    int gpu_ids[2] = {0, 1}; // weirdly titan x gpus are listed as indices 1 and 3 in nvidia-smi but are actually indices 0 and 1

    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    if (gpu_ids[0] >= gpu_count || gpu_ids[1] >= gpu_count) {
        printf("Requested GPUs not available\n");
        return;
    }

    hipDeviceProp_t prop1, prop3;
    hipGetDeviceProperties(&prop1, gpu_ids[0]);
    hipGetDeviceProperties(&prop3, gpu_ids[1]);
    printf("Using GPUs: %d (%s) and %d (%s)\n", 
           gpu_ids[0], prop1.name, 
           gpu_ids[1], prop3.name);

    hipEvent_t start[2], stop[2], overall_start, overall_stop;
    hipStream_t streams[2];
    
    for (int gpu = 0; gpu < 2; gpu++) {
        hipSetDevice(gpu_ids[gpu]);
        hipEventCreate(&start[gpu]);
        hipEventCreate(&stop[gpu]);
        hipStreamCreate(&streams[gpu]);
    }
    hipEventCreate(&overall_start);
    hipEventCreate(&overall_stop);
    hipEventRecord(overall_start, 0);

    int iter = 0;
    int * count = new int[k];
    // double acc_red_time = 0.0f, acc_assign_time = 0.0f, acc_update_time = 0.0f, acc_centroid_time = 0.0f;
    double acc_assign_time[2] = {0.0f, 0.0f};

    int n_per_gpu = n / 2;
    int remainder = n % 2;
    int n_gpu0 = n_per_gpu + remainder;
    int n_gpu1 = n_per_gpu;

    int *d_x[2], *d_y[2], *d_c[2];
    double *d_cx[2], *d_cy[2], *d_sumx[2], *d_sumy[2];
    int *d_count[2];
    bool cont = false;

    for (int gpu = 0; gpu < 2; gpu++) {
        hipSetDevice(gpu_ids[gpu]);
        int current_n = (gpu == 0) ? n_gpu0 : n_gpu1;
        
        hipMalloc(&d_x[gpu], current_n * sizeof(int));
        hipMalloc(&d_y[gpu], current_n * sizeof(int));
        hipMalloc(&d_c[gpu], current_n * sizeof(int));
        hipMalloc(&d_cx[gpu], k * sizeof(double));
        hipMalloc(&d_cy[gpu], k * sizeof(double));
        hipMalloc(&d_sumx[gpu], k * sizeof(double));
        hipMalloc(&d_sumy[gpu], k * sizeof(double));
        hipMalloc(&d_count[gpu], k * sizeof(int));
    }

    hipSetDevice(gpu_ids[0]);
    hipMemcpy(d_x[0], x, n_gpu0 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y[0], y, n_gpu0 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cx[0], cx, k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_cy[0], cy, k * sizeof(double), hipMemcpyHostToDevice);

    hipSetDevice(gpu_ids[1]);
    hipMemcpy(d_x[1], x + n_gpu0, n_gpu1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y[1], y + n_gpu0, n_gpu1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cx[1], cx, k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_cy[1], cy, k * sizeof(double), hipMemcpyHostToDevice);

    printf("Begin\n");

    int blockSize = 256;
    int gridSize[2];
    gridSize[0] = (n_gpu0 + blockSize - 1) / blockSize;
    gridSize[1] = (n_gpu1 + blockSize - 1) / blockSize;

    for (int gpu = 0; gpu < 2; gpu++) {
        hipSetDevice(gpu_ids[gpu]);
        hipMemset(d_c[gpu], -1, (gpu == 0 ? n_gpu0 : n_gpu1) * sizeof(int));
    }
    #ifdef FLAT
    if (k > 1024){
        printf("Cannot use flat assignment for k > 1024, using regular assignment instead\n");
    }
    #endif

    while (iter < MAXITER) {
        cont = false;
        
        for (int gpu = 0; gpu < 2; gpu++) {
            hipSetDevice(gpu_ids[gpu]);
            hipMemset(d_sumx[gpu], 0, k * sizeof(double));
            hipMemset(d_sumy[gpu], 0, k * sizeof(double));
            hipMemset(d_count[gpu], 0, k * sizeof(int));
        }

        #ifdef FLAT
            int assing_gridsize[2];
            int assign_blockSize = k;
            assing_gridsize[0] = ((n_gpu0 * k) + assign_blockSize - 1)/assign_blockSize;
            assing_gridsize[1] = ((n_gpu1 * k) + assign_blockSize - 1)/assign_blockSize;
        #endif

        for (int gpu = 0; gpu < 2; gpu++) {
            hipSetDevice(gpu_ids[gpu]);
            int current_n = (gpu == 0) ? n_gpu0 : n_gpu1;
            
            hipEventRecord(start[gpu], streams[gpu]);
            #ifdef FLAT
            if (k > 1024){
                assignKernel<<<gridSize[gpu], blockSize, 0, streams[gpu]>>>( d_x[gpu], d_y[gpu], d_c[gpu], d_cx[gpu], d_cy[gpu], k, current_n);
            } else {
                assignKernel_flat<<<assing_gridsize[gpu], assign_blockSize, k*sizeof(shared_min_t), streams[gpu]>>>( d_x[gpu], d_y[gpu], d_c[gpu], d_cx[gpu], d_cy[gpu], k, current_n);
            }
            #else
                assignKernel<<<gridSize[gpu], blockSize, 0, streams[gpu]>>>( d_x[gpu], d_y[gpu], d_c[gpu], d_cx[gpu], d_cy[gpu], k, current_n);
            #endif
            hipDeviceSynchronize();
            hipEventRecord(stop[gpu], streams[gpu]);
            hipEventSynchronize(stop[gpu]);
            float timer_assign = 0.0f;
            hipEventElapsedTime(&timer_assign, start[gpu], stop[gpu]);
            acc_assign_time[gpu] += timer_assign;
            
            updateKernel<<<gridSize[gpu], blockSize, 0, streams[gpu]>>>(
                d_x[gpu], d_y[gpu], d_c[gpu], k, current_n, d_sumx[gpu], d_sumy[gpu], d_count[gpu]
            );
        }

        for (int gpu = 0; gpu < 2; gpu++) {
            hipSetDevice(gpu_ids[gpu]);
            hipDeviceSynchronize();
        }

        hipSetDevice(gpu_ids[0]);
        double *h_sumx = new double[k];
        double *h_sumy = new double[k];
        int *h_count = new int[k];

        hipMemcpy(h_sumx, d_sumx[0], k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_sumy, d_sumy[0], k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_count, d_count[0], k * sizeof(int), hipMemcpyDeviceToHost);

        double *h_sumx1 = new double[k];
        double *h_sumy1 = new double[k];
        int *h_count1 = new int[k];

        hipSetDevice(gpu_ids[1]);
        hipMemcpy(h_sumx1, d_sumx[1], k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_sumy1, d_sumy[1], k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_count1, d_count[1], k * sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < k; i++) {
            h_sumx[i] += h_sumx1[i];
            h_sumy[i] += h_sumy1[i];
            h_count[i] += h_count1[i];
        }

        for (int gpu = 0; gpu < 2; gpu++) {
            hipSetDevice(gpu_ids[gpu]);
            hipMemcpy(d_sumx[gpu], h_sumx, k * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_sumy[gpu], h_sumy, k * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_count[gpu], h_count, k * sizeof(int), hipMemcpyHostToDevice);

            computeCentroids<<<(k + blockSize - 1) / blockSize, blockSize, 0, streams[gpu]>>>(
                k, d_sumx[gpu], d_sumy[gpu], d_count[gpu], d_cx[gpu], d_cy[gpu]
            );
        }

        double *prev_cx = new double[k];
        double *prev_cy = new double[k];
        memcpy(prev_cx, cx, k * sizeof(double));
        memcpy(prev_cy, cy, k * sizeof(double));

        hipSetDevice(gpu_ids[0]);
        hipMemcpy(cx, d_cx[0], k * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(cy, d_cy[0], k * sizeof(double), hipMemcpyDeviceToHost);

        for (int i = 0; i < k; i++) {
            cont |= ((cx[i] != prev_cx[i]) || (cy[i] != prev_cy[i]));
        }

        delete[] h_sumx;
        delete[] h_sumy;
        delete[] h_count;
        delete[] h_sumx1;
        delete[] h_sumy1;
        delete[] h_count1;
        delete[] prev_cx;
        delete[] prev_cy;

        if (!cont) {
            hipSetDevice(gpu_ids[0]);
            hipMemcpy(c, d_c[0], n_gpu0 * sizeof(int), hipMemcpyDeviceToHost);
            hipSetDevice(gpu_ids[1]);
            hipMemcpy(c + n_gpu0, d_c[1], n_gpu1 * sizeof(int), hipMemcpyDeviceToHost);
            
            printf("Converged at iteration %d\n", iter);
            writeClusterAssignments(x, y, c, n, OUTFILE);
            break;
        }

        iter++;
    }

    for (int gpu = 0; gpu < 2; gpu++) {
        hipSetDevice(gpu_ids[gpu]);
        hipFree(d_x[gpu]);
        hipFree(d_y[gpu]);
        hipFree(d_c[gpu]);
        hipFree(d_cx[gpu]);
        hipFree(d_cy[gpu]);
        hipFree(d_sumx[gpu]);
        hipFree(d_sumy[gpu]);
        hipFree(d_count[gpu]);
        hipEventDestroy(start[gpu]);
        hipEventDestroy(stop[gpu]);
        hipStreamDestroy(streams[gpu]);
    }

    hipEventRecord(overall_stop, 0);
    hipEventSynchronize(overall_stop);
    float timer_overall;
    hipEventElapsedTime(&timer_overall, overall_start, overall_stop);
    printf("Accumulated assign kernel runtime gpu 1: %f seconds\n", acc_assign_time[0]/1000);
    printf("Accumulated assign kernel runtime gpu 2: %f seconds\n", acc_assign_time[1]/1000);
    printf("Kmeans total runtime (Cuda events): %f seconds\n", timer_overall/1000);
}

int readfile(const string& fname, int*& x, int*& y) {
    ifstream f;
    f.open(fname.c_str());
    if (!f.is_open()) {
        cerr << "Error opening file: " << fname << endl;
        exit(-1);
    }

    string line;
    getline(f, line);  // Read the first line (number of points)
    int n = atoi(line.c_str());
    #ifdef DEBUG
        printf("file length: %d\n", n);
    #endif

    #ifdef ALIGNED_ALLOC
        x = (int*) aligned_alloc(64, n * sizeof(int));
        y = (int*) aligned_alloc(64, n * sizeof(int));
    #else
        x = new int[n];
        y = new int[n];
    #endif

    int tempx, tempy;
    for (int i = 0; i < n; i++) {
        getline(f, line);
        stringstream ss(line);
        ss >> tempx >> tempy;
        x[i] = tempx;
        y[i] = tempy;
    }

    return n;
}


int main(int argc, char *argv[]) {
    if (argc - 1 != 2) {
        printf("./kmeans <filename> <k>\n");
        exit(-1);
    } else if (atoi(argv[2]) < 1){
        printf("k must be a positive integer number\n");
        exit(-1);
    }

    string fname = argv[1];
    int k = atoi(argv[2]);
    
    int *x, *y, *c;
    double *cx, *cy;

    int n = readfile(fname, x, y);
    
    cx = new double[k];
    cy = new double[k];
    c = new int[n];

    double init_begin = omp_get_wtime();
    randomCenters(x, y, n, k, cx, cy);
    double init_end = omp_get_wtime();
    printf("Random Centers Init Time (OMP wtime): %f seconds \n", init_end - init_begin);

    double kmeans_start = omp_get_wtime();
    #ifndef MULTIGPU
        kmeans(x, y, c, cx, cy, k, n);
    #else
        printf("Multi GPU\n");
        kmeans_multigpu(x, y, c, cx, cy, k, n);
    #endif

    double kmeans_end = omp_get_wtime();
    printf("K-Means Execution Time (OMP wtime): %f seconds \n", kmeans_end - kmeans_start);

    #ifdef DEBUG
        for (int i = 0; i < k; i++){
            cout << "cluster " << i << " " << cx[i] << ", " << cy[i] << endl;
        }
    #endif

    double totalSSD = 0.0;
    for (int i = 0; i < n; i++) {
        int cluster = c[i];
        totalSSD += dist(x[i], y[i], cx[cluster], cy[cluster]);
    }

    printf("Sqrt of Sum of Squared Distances (SSD): %f\n", sqrt(totalSSD));

    delete[] x;
    delete[] y;
    delete[] cx;
    delete[] cy;
    delete[] c;

    return 0;
}
